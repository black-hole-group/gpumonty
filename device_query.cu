#include "hip/hip_runtime.h"
#include "decs.h"    
#include "harm_model.h"

/*
Queries GPU for the amount of free memory.
Returns max number of photons that GPU can hold at once.
If number of photons desired is smaller than nmaxgpu, then
returns total number of photons desired instead.

in: size of HARM arrays along each dimension
out: max number of superphotons GPU can hold at once
 
The GPU must hold the following n1xn2xn3 arrays in its 
global memory during processing: 3*B, rho, T, 4*v
 */
int get_max_photons(int n1, int n2, int n3) {
    hipDeviceProp_t prop;
    size_t free, total;
    
    // assumes one GPU
    hipGetDeviceProperties(&prop, 0); 
    hipMemGetInfo(&free, &total);

    float memtotal=(float)prop.totalGlobalMem; // bytes
    float memfree=(float)free; // bytes

    // total size of HARM arrays in bytes, 9+1 arrays
    // adds one extra array to leave room and avoid filling up the memory
    float sizeHARM=10.0*n1*n2*n3;

    // estimates max number of photons GPU can process at once
    // based on size of of_photon struct
    int nmax=(int)((memfree-sizeHARM)/(25.*4.));

    printf("GPU model: %s\n", prop.name);
    printf("Total memory = %f (GB)\n", memtotal/1e9);
    printf("Free memory = %f (GB)\n", memfree/1e9);
    printf("Max superphotons processed each pass = %d\n", nmax);    

    return nmax;
}





/*
Transfer HARM arrays to GPU.

TODO:
- [ ] modify p to include extra spatial dimension
*/
void mallocDevice(int nprim, int n1, int n2, int n3) {
    hipMalloc(&d_p, nprim*n1*n2*n3*sizeof(float));
    hipMemcpy(d_p, p, nprim*n1*n2*n3*sizeof(float), hipMemcpyHostToDevice);
}




/*

Deallocates device variables

*/
void freeDevice() {
    hipFree(d_p);
}