#include "hip/hip_runtime.h"
/*
Queries GPU for the amount of free memory.
Returns max number of photons that GPU can hold at once.

in: size of HARM arrays along each dimension
out: max number of superphotons GPU can hold at once

The GPU must hold the following n1xn2xn3 arrays in its 
global memory during processing: 3*B, rho, T, 4*v
*/
#include "device_query.h" 

int get_max_photons(int n1, int n2, int n3) {
    hipDeviceProp_t prop;
    size_t free, total;
    
    // assumes one GPU
    hipGetDeviceProperties(&prop, 0); 
    hipMemGetInfo(&free, &total);

    float memtotal=(float)prop.totalGlobalMem; // bytes
    float memfree=(float)free; // bytes

    // total size of HARM arrays in bytes, 9+1 arrays
    // adds one extra array to leave room and avoid filling up the memory
    float sizeHARM=10.0*n1*n2*n3;

    // estimates max number of photons GPU can process at once
    // based on size of of_photon struct
    int nmax=(int)((memfree-sizeHARM)/(25.*4.));

    printf("GPU model: %s\n", prop.name);
    printf("Total memory = %f (GB)\n", memtotal/1e9);
    printf("Free memory = %f (GB)\n", memfree/1e9);
    printf("Max superphotons processed each pass = %d\n", nmax);    

    return nmax;
}