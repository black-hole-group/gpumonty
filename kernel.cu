#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel.h"    
#define TPB 32 // number of threads per block 



__global__ 
void testKernel(double *d_p, int nprim, int n1, int n2, double *d_pharr, int nph, int nphvars)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nph) return;

	// shows X0 for each photon
	printf("photon[%d]=%lf\n", i,d_pharr[i*nphvars+2]);
}



void launchKernel(double *p, int nprim, int n1, int n2, double *pharr, int nph, int nphvars) 
{
	// device variables
	double *d_p=0; // HARM arrays
	double *d_pharr=0; // superphoton array

	// send HARM arrays to device
    hipMalloc(&d_p, nprim*n1*n2*sizeof(double));
    hipMemcpy(d_p, p, nprim*n1*n2*sizeof(double), hipMemcpyHostToDevice);

    // send photon initial conditions to device
    hipMalloc(&d_pharr, nphvars*nph*sizeof(double));
    hipMemcpy(d_pharr, pharr, nphvars*nph*sizeof(double), hipMemcpyHostToDevice);

	testKernel<<<(nph+TPB-1)/TPB, TPB>>>(d_p, nprim, n1, n2, d_pharr, nph, nphvars);

	// frees device memory
	hipFree(d_p);
	hipFree(d_pharr);
}