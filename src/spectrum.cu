#include "hip/hip_runtime.h"
#include "decs.h"
#include "harm_model.h"
#include <hip/hip_runtime.h>
#include "gpu_utils.h"

#define SPECTRUM_FILE_NAME	"grmonty.spec"

unsigned long long N_superph_recorded = 0;
struct of_spectrum spect[N_THBINS][N_EBINS];

__device__ unsigned long long d_N_superph_recorded = 0;
__device__ struct of_spectrum d_spect[N_THBINS][N_EBINS];


/*******************************************************************************
* Host-only Functions
*
*******************************************************************************/

void init_spectrum () {
	for (int i = 0; i < N_THBINS; i++)
		for (int j = 0; j < N_EBINS; j++) {
			spect[i][j].dNdlE = 0.0;
			spect[i][j].dEdlE = 0.0;
			spect[i][j].nph = 0.0;
			spect[i][j].nscatt = 0.0;
			spect[i][j].X1iav = 0.0;
			spect[i][j].X2isq = 0.0;
			spect[i][j].X3fsq = 0.0;
			spect[i][j].tau_abs = 0.0;
			spect[i][j].tau_scatt = 0.0;
			spect[i][j].ne0 = 0.0;
			spect[i][j].thetae0 = 0.0;
			spect[i][j].b0 = 0.0;
			spect[i][j].E0 = 0.0;;
		}

	CUDASAFE(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_spect), spect,
				    N_THBINS*N_EBINS*sizeof(struct of_spectrum),
				    0, hipMemcpyHostToDevice));
}

void copy_spect_from_gpu() {
	CUDASAFE(hipMemcpyFromSymbol(spect, HIP_SYMBOL(d_spect),
				      N_THBINS*N_EBINS*sizeof(struct of_spectrum),
				      0, hipMemcpyDeviceToHost));
	CUDASAFE(hipMemcpyFromSymbol(&N_superph_recorded, HIP_SYMBOL(d_N_superph_recorded),
				      sizeof(unsigned long long), 0,
				      hipMemcpyDeviceToHost));
}

/* output spectrum to file */
void report_spectrum(unsigned long long N_superph_made)
{
	int i, j;
	double dx2, dOmega, nuLnu, tau_scatt, L;
	FILE *fp;

	double nu0,nu1,nu,fnu ;
	double dsource = 8000*PC ;

	fp = fopen(SPECTRUM_FILE_NAME, "w");
	if (fp == NULL) {
		fprintf(stderr, "trouble opening spectrum file\n");
		exit(0);
	}

	/* output */
	max_tau_scatt = 0.;
	L = 0.;
	for (i = 0; i < N_EBINS; i++) {

		/* output log_10(photon energy/(me c^2)) */
		fprintf(fp, "%10.5g ", (i * dlE + lE0) / M_LN10);

		for (j = 0; j < N_THBINS; j++) {

			/* convert accumulated photon number in each bin
			   to \nu L_\nu, in units of Lsun */
			dx2 = (stopx[2] - startx[2]) / (2. * N_THBINS);

			/* factor of 2 accounts for folding around equator */
			dOmega = 2. * dOmega_func(j * dx2, (j + 1) * dx2);

			nuLnu =
			    (ME * CL * CL) * (4. * M_PI / dOmega) * (1. /
								     dlE);

			nuLnu *= spect[j][i].dEdlE;
			nuLnu /= LSUN;

			tau_scatt =
			    spect[j][i].tau_scatt / (spect[j][i].dNdlE +
						     SMALL);
			fprintf(fp,
				"%10.5g %10.5g %10.5g %10.5g %10.5g %10.5g ",
				nuLnu,
				spect[j][i].tau_abs / (spect[j][i].dNdlE +
						       SMALL), tau_scatt,
				spect[j][i].X1iav / (spect[j][i].dNdlE +
						     SMALL),
				sqrt(fabs
				     (spect[j][i].X2isq /
				      (spect[j][i].dNdlE + SMALL))),
				sqrt(fabs
				     (spect[j][i].X3fsq /
				      (spect[j][i].dNdlE + SMALL)))
			    );


			nu0 = ME * CL * CL * exp((i - 0.5) * dlE + lE0) / HPL ;
			nu1 = ME * CL * CL * exp((i + 0.5) * dlE + lE0) / HPL ;

			if(nu0 < 230.e9 && nu1 > 230.e9) {
				nu = ME * CL * CL * exp(i * dlE + lE0) / HPL ;
				fnu = nuLnu*LSUN/(4.*M_PI*dsource*dsource*nu*JY) ;
				fprintf(stderr,"fnu: %10.5g\n",fnu) ;
			}

			/* added to give average # scatterings */
			fprintf(fp,"%10.5g ",spect[j][i].nscatt/ (
				spect[j][i].dNdlE + SMALL)) ;

			if (tau_scatt > max_tau_scatt)
				max_tau_scatt = tau_scatt;

			L += nuLnu * dOmega * dlE / (4. * M_PI);
		}
		fprintf(fp, "\n");
	}
	fprintf(stderr,
		"luminosity %g, dMact %g, efficiency %g, L/Ladv %g, max_tau_scatt %g\n",
		L, dMact * M_unit / T_unit / (MSUN / YEAR),
		L * LSUN / (dMact * M_unit * CL * CL / T_unit),
		L * LSUN / (Ladv * M_unit * CL * CL / T_unit),
		max_tau_scatt);
	fprintf(stderr, "\n");
	fprintf(stderr, "N_superph_made: %llu\n", N_superph_made);
	fprintf(stderr, "N_superph_recorded: %llu\n", N_superph_recorded);

	fclose(fp);

}


/*******************************************************************************
* Device-only Functions
*
*******************************************************************************/

/*
	record contribution of super photon to spectrum.

	This routine should make minimal assumptions about the
	coordinate system.
*/
__device__
void record_super_photon(struct of_photon *ph)
{
	double lE, dx2;
	int iE, ix2;

	if (isnan(ph->w) || isnan(ph->E)) {
		// fprintf(stderr, "record isnan: %g %g\n", ph->w, ph->E);
		return;
	}
// #pragma omp critical (MAXTAU)
// 	{
// 		if (ph->tau_scatt > max_tau_scatt)
// 			max_tau_scatt = ph->tau_scatt;
// 	}
	// atomicMax(&max_tau_scatt, d2i(ph->tau_scatt));
	/* currently, bin in x2 coordinate */

	/* get theta bin, while folding around equator */
	dx2 = (d_stopx[2] - d_startx[2]) / (2. * N_THBINS);
	if (ph->X[2] < 0.5 * (d_startx[2] + d_stopx[2]))
		ix2 = (int) (ph->X[2] / dx2);
	else
		ix2 = (int) ((d_stopx[2] - ph->X[2]) / dx2);

	/* check limits */
	if (ix2 < 0 || ix2 >= N_THBINS)
		return;

	/* get energy bin */
	lE = log(ph->E);
	iE = (int) ((lE - lE0) / dlE + 2.5) - 2;	/* bin is centered on iE*dlE + lE0 */

	/* check limits */
	if (iE < 0 || iE >= N_EBINS)
		return;

	atomicAdd(&d_N_superph_recorded, 1);
	// #atomic
	// N_scatt += ph->nscatt;

	/* sum in photon */
	atomicAdd(&d_spect[ix2][iE].dNdlE, ph->w);
	atomicAdd(&d_spect[ix2][iE].dEdlE, ph->w * ph->E);
	atomicAdd(&d_spect[ix2][iE].tau_abs, ph->w * ph->tau_abs);
	atomicAdd(&d_spect[ix2][iE].tau_scatt, ph->w * ph->tau_scatt);
	atomicAdd(&d_spect[ix2][iE].X1iav, ph->w * ph->X1i);
	atomicAdd(&d_spect[ix2][iE].X2isq, ph->w * (ph->X2i * ph->X2i));
	atomicAdd(&d_spect[ix2][iE].X3fsq, ph->w * (ph->X[3] * ph->X[3]));
	atomicAdd(&d_spect[ix2][iE].ne0, ph->w * (ph->ne0));
	atomicAdd(&d_spect[ix2][iE].b0, ph->w * (ph->b0));
	atomicAdd(&d_spect[ix2][iE].thetae0, ph->w * (ph->thetae0));
	atomicAdd(&d_spect[ix2][iE].nscatt, ph->w * ph->nscatt);
	atomicAdd(&d_spect[ix2][iE].nph, 1.);

}
