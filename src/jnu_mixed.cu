#include "hip/hip_runtime.h"


#include "decs.h"
#include "bessel.h"
#include <hip/hip_runtime.h>
#include "gpu_utils.h"
/*

"mixed" emissivity formula

interpolates between Petrosian limit and
classical thermal synchrotron limit

good for Thetae > 1

*/

static double F[N_ESAMP + 1];
static double lK_min, dlK;
static double K2[N_ESAMP + 1];
static double lT_min, j_dlT;

__device__ static double d_K2[N_ESAMP + 1];
__device__ static double d_lT_min, d_j_dlT;


static double linear_interp_F(double K);
__host__ __device__
static double linear_interp_K2(double Thetae);


/*******************************************************************************
* Host-only Functions
*
*******************************************************************************/

#define JCST (M_SQRT2*EE*EE*EE/(27*ME*CL*CL))
double int_jnu(double Ne, double Thetae, double Bmag, double nu)
{
/* Returns energy per unit time at							*
 * frequency nu in cgs										*/

	double j_fac, l_K2;

	if (Thetae < THETAE_MIN)
		return 0.;

	l_K2 = K2_eval(Thetae);
	if (l_K2 == 0.)
		return 0.;

	j_fac = Ne * Bmag * Thetae * Thetae / l_K2;

	return JCST * j_fac * F_eval(Thetae, Bmag, nu);
}
#undef JCST

#define CST (1.88774862536) /* 2^{11/12} */
double jnu_integrand(double th, void *params)
{

	double K = *(double *) params;
	double sth = sin(th);
	double x = K / sth;

	if (sth < 1.e-150 || x > 2.e8)
		return 0.;

	return sth * sth * pow(sqrt(x) + CST * pow(x, 1. / 6.),
			       2.) * exp(-pow(x, 1. / 3.));
}
#undef CST

static double linear_interp_F(double K)
{

	int i;
	double di, lK;

	lK = log(K);

	di = (lK - lK_min) * dlK;
	i = (int) di;
	di = di - i;

	return exp((1. - di) * F[i] + di * F[i + 1]);
}

#define EPSABS (0.)
#define EPSREL (1.e-6)
#define KMIN (0.002)
#define KMAX (1.e7)
#define TMIN (THETAE_MIN)
#define TMAX (1.e2)
void init_emiss_tables(void)
{

	int k;
	double result, err, K, T;
	gsl_function func;
	gsl_integration_workspace *w;

	func.function = &jnu_integrand;
	func.params = &K;

	lK_min = log(KMIN);
	dlK = log(KMAX / KMIN) / (N_ESAMP);

	lT_min = log(TMIN);
	j_dlT = log(TMAX / TMIN) / (N_ESAMP);

	/*  build table for F(K) where F(K) is given by
	   \int_0^\pi ( (K/\sin\theta)^{1/2} + 2^{11/12}(K/\sin\theta)^{1/6})^2 \exp[-(K/\sin\theta)^{1/3}]
	   so that J_{\nu} = const.*F(K)
	 */
	w = gsl_integration_workspace_alloc(1000);
	for (k = 0; k <= N_ESAMP; k++) {
		K = exp(k * dlK + lK_min);
		gsl_integration_qag(&func, 0., M_PI / 2., EPSABS, EPSREL,
				    1000, GSL_INTEG_GAUSS61, w, &result,
				    &err);
		F[k] = log(4 * M_PI * result);
	}
	gsl_integration_workspace_free(w);

	/*  build table for quick evaluation of the bessel function K2 for emissivity */
	for (k = 0; k <= N_ESAMP; k++) {
		T = exp(k * j_dlT + lT_min);
		K2[k] = log(bessk(2, 1. / T));

	}

	/* Avoid doing divisions later */
	dlK = 1. / dlK;
	j_dlT = 1. / j_dlT;

	CUDASAFE(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_K2), K2, (N_ESAMP+1)*sizeof(double), 0, hipMemcpyHostToDevice));
	CUDASAFE(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_lT_min), &lT_min, sizeof(double), 0, hipMemcpyHostToDevice));
	CUDASAFE(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_j_dlT), &j_dlT, sizeof(double), 0, hipMemcpyHostToDevice));

	fprintf(stderr, "done.\n\n");

	return;
}

#define KFAC	(9*M_PI*ME*CL/EE)
double F_eval(double Thetae, double Bmag, double nu)
{

	double K, x;

	K = KFAC * nu / (Bmag * Thetae * Thetae);

	if (K > KMAX) {
		return 0.;
	} else if (K < KMIN) {
		/* use a good approximation */
		x = pow(K, 0.333333333333333333);
		return (x * (37.67503800178 + 2.240274341836 * x));
	} else {
		return linear_interp_F(K);
	}
}
#undef KFAC

/*******************************************************************************
* Host/Device Functions
*
*******************************************************************************/

/* rapid evaluation of K_2(1/\Thetae) */
__host__ __device__
double K2_eval(double Thetae)
{
	if (Thetae < THETAE_MIN)
		return 0.;
	if (Thetae > TMAX)
		return 2. * Thetae * Thetae;

	return linear_interp_K2(Thetae);
}

#undef KMIN
#undef KMAX
#undef EPSABS
#undef EPSREL

#define CST 1.88774862536	/* 2^{11/12} */
__host__ __device__
double jnu_synch(double nu, double Ne, double Thetae, double B,
		 double theta)
{
	double l_K2, nuc, nus, x, f, j, sth, xp1, xx;

	if (Thetae < THETAE_MIN)
		return 0.;

	l_K2 = K2_eval(Thetae);

	nuc = EE * B / (2. * M_PI * ME * CL);
	sth = sin(theta);
	nus = (2. / 9.) * nuc * Thetae * Thetae * sth;
	if (nu > 1.e12 * nus)
		return (0.);
	x = nu / nus;
	xp1 = pow(x, 1. / 3.);
	xx = sqrt(x) + CST * sqrt(xp1);
	f = xx * xx;
	j = (M_SQRT2 * M_PI * EE * EE * Ne * nus / (3. * CL * l_K2)) * f *
	    exp(-xp1);

	return (j);
}
#undef CST

__host__ __device__
static double linear_interp_K2(double Thetae)
{

#ifdef __CUDA_ARCH__
	#define AS_K2 d_K2
	#define AS_j_dlT d_j_dlT
	#define AS_lT_min d_lT_min
#else
	#define AS_K2 K2
	#define AS_j_dlT j_dlT
	#define AS_lT_min lT_min
#endif

	int i;
	double di, lT;

	lT = log(Thetae);

	di = (lT - AS_lT_min) * AS_j_dlT;
	i = (int) di;
	di = di - i;
	return exp((1. - di) * AS_K2[i] + di * AS_K2[i + 1]);

#undef AS_K2
#undef AS_j_dlT
#undef AS_lT_min
}
