#include "hip/hip_runtime.h"

/***********************************************************************************
    Copyright 2013 Joshua C. Dolence, Charles F. Gammie, Monika Mo\'scibrodzka,
                   and Po Kin Leung

                        GRMONTY  version 1.0   (released February 1, 2013)

    This file is part of GRMONTY.  GRMONTY v1.0 is a program that calculates the
    emergent spectrum from a model using a Monte Carlo technique.

    This version of GRMONTY is configured to use input files from the HARM code
    available on the same site.   It assumes that the source is a plasma near a
    black hole described by Kerr-Schild coordinates that radiates via thermal
    synchrotron and inverse compton scattering.

    You are morally obligated to cite the following paper in any
    scientific literature that results from use of any part of GRMONTY:

    Dolence, J.C., Gammie, C.F., Mo\'scibrodzka, M., \& Leung, P.-K. 2009,
        Astrophysical Journal Supplement, 184, 387

    Further, we strongly encourage you to obtain the latest version of
    GRMONTY directly from our distribution website:
    http://rainman.astro.illinois.edu/codelib/

    GRMONTY is free software; you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation; either version 2 of the License, or
    (at your option) any later version.

    GRMONTY is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GRMONTY; if not, write to the Free Software
    Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA  02110-1301  USA

***********************************************************************************/


/*

model-independent radiation-related utilities.

*/

#include "decs.h"

__device__ double Bnu_inv(double nu, double Thetae)
{

	double x;

	x = HPL * nu / (ME * CL * CL * Thetae);

	if (x < 1.e-3)		/* Taylor expand */
		return ((2. * HPL / (CL * CL)) /
			(x / 24. * (24. + x * (12. + x * (4. + x)))));
	else
		return ((2. * HPL / (CL * CL)) / (exp(x) - 1.));
}

__device__ double jnu_inv(double nu, double Thetae, double Ne, double B, double theta)
{
	double j;

	j = jnu_synch(nu, Ne, Thetae, B, theta);

	return (j / (nu * nu));
}

/* return Lorentz invariant scattering opacity */
__device__ double alpha_inv_scatt(double nu, double Thetae, double Ne)
{
	double kappa;

	kappa = kappa_es(nu, Thetae);

	return (nu * kappa * Ne * MP);
}

/* return Lorentz invariant absorption opacity */
__device__ double alpha_inv_abs(double nu, double Thetae, double Ne, double B,
		     double theta)
{
	double j, bnu;

	j = jnu_inv(nu, Thetae, Ne, B, theta);
	bnu = Bnu_inv(nu, Thetae);

	return (j / (bnu + 1.e-100));
}


/* return electron scattering opacity, in cgs */
__device__ double kappa_es(double nu, double Thetae)
{
	double Eg;

	/* assume pure hydrogen gas to
	   convert cross section to opacity */
	Eg = HPL * nu / (ME * CL * CL);
	return (total_compton_cross_lkup(Eg, Thetae) / MP);
}

/* get frequency in fluid frame, in Hz */
__device__ double get_fluid_nu(double X[4], double K[4], double Ucov[NDIM])
{
	double ener, nu;

	/* this is the energy in electron rest-mass units */
	ener = -(K[0] * Ucov[0] +
		 K[1] * Ucov[1] + K[2] * Ucov[2] + K[3] * Ucov[3]);

	nu = ener * ME * CL * CL / HPL;

	if (isnan(ener)) {
		printf("isnan get_fluid_nu, K: %g %g %g %g\n",
			K[0], K[1], K[2], K[3]);
		printf("isnan get_fluid_nu, X: %g %g %g %g\n",
			X[0], X[1], X[2], X[3]);
		printf("isnan get_fluid_nu, U: %g %g %g %g\n",
			Ucov[0], Ucov[1], Ucov[2], Ucov[3]);
	}
	return nu;
}

/* return angle between magnetic field and wavevector */
__device__ double get_bk_angle(
	double X[NDIM],
	double K[NDIM],
	double Ucov[NDIM],
	double Bcov[NDIM],
	double B)
{

	double k, mu;

	if (B == 0.)
		return (M_PI / 2.);

	k = fabs(K[0] * Ucov[0] + K[1] * Ucov[1] + K[2] * Ucov[2] +
		 K[3] * Ucov[3]);

	/* B is in cgs but Bcov is in code units */
	mu = (K[0] * Bcov[0] + K[1] * Bcov[1] + K[2] * Bcov[2] +
	      K[3] * Bcov[3]) / (k * B / B_unit);

	if (fabs(mu) > 1.)
		mu /= fabs(mu);

	return (acos(mu));
}
